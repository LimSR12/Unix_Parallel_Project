
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(){
	printf("Hello from GPU - thread %d\n", threadIdx.x);
}

int main(){
	helloFromGPU<<<1, 10>>>();
	hipDeviceSynchronize();
	return 0;
}
